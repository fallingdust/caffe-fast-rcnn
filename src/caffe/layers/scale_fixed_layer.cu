#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/scale_fixed_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ScaleForward(const int n, const Dtype* in,
    const Dtype* scale, const int scale_dim, const int inner_dim,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int scale_index = (index / inner_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index];
  }
}

template <typename Dtype>
__global__ void ScaleBiasForward(const int n, const Dtype* in,
    const Dtype* scale, const Dtype* bias,
    const int scale_dim, const int inner_dim, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int scale_index = (index / inner_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index] + bias[scale_index];
  }
}

template <typename Dtype>
void ScaleFixedLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* scale_data = this->blobs_[0].get()->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  if (bias_layer_) {
    const Dtype* bias_data = this->blobs_[this->blobs_.size() - 1]->gpu_data();
    ScaleBiasForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, scale_data, bias_data, scale_dim_, inner_dim_,
        top_data);
  } else {
    ScaleForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, scale_data, scale_dim_, inner_dim_, top_data);
  }
}

template <typename Dtype>
void ScaleFixedLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Blob<Dtype>* scale = this->blobs_[0].get();
  if (propagate_down[0]) {
    const int count = top[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* scale_data = scale->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    ScaleForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, scale_data, scale_dim_, inner_dim_, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ScaleFixedLayer);

}  // namespace caffe
