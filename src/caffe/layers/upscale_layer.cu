#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/upscale_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void UpscaleForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int bottom_height, const int bottom_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * bottom_height * bottom_width;
    const int bh = int(h * bottom_height / float(height));
    const int hw = int(w * bottom_width / float(width));
    top_data[index] = bottom_slice[bh * bottom_width + hw];
  }
}


template <typename Dtype>
void UpscaleingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  UpscaleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top[0]->num(), top[0]->channels(),
      top[0]->height(), top[0]->width(), bottom[0]->height(), bottom[0]->width(), top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void UpscaleingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}


INSTANTIATE_LAYER_GPU_FUNCS(UpscaleingLayer);


}  // namespace caffe
