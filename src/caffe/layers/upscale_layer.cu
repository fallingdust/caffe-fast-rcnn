#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/upscale_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void UpscaleForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int bottom_height, const int bottom_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * bottom_height * bottom_width;
    const int bh = int(h * bottom_height / float(height));
    const int hw = int(w * bottom_width / float(width));
    top_data[index] = bottom_slice[bh * bottom_width + hw];
  }
}

template <typename Dtype>
__global__ void UpscaleBackward(const int nthreads,
    const Dtype* const top_diff, const int num, const int channels,
    const int height, const int width, const int bottom_height, const int bottom_width, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    Dtype* bottom_slice =
        bottom_diff + (n * channels + c) * bottom_height * bottom_width;
    const int bh = int(h * bottom_height / float(height));
    const int hw = int(w * bottom_width / float(width));
    bottom_slice[bh * bottom_width + hw] += top_diff[index];
  }
}

template <typename Dtype>
void UpscaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  UpscaleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top[0]->num(), top[0]->channels(),
      top[0]->height(), top[0]->width(), bottom[0]->height(), bottom[0]->width(), top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void UpscaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->cpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_cpu_diff();
  int count = top[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // NOLINT_NEXT_LINE(whitespace/operators)
  UpscaleBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, top[0]->num(), top[0]->channels(),
      top[0]->height(), top[0]->width(), bottom[0]->height(), bottom[0]->width(), bottom_diff);

  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UpscaleLayer);


}  // namespace caffe
