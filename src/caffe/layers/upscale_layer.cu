#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/upscale_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void UpscaleForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int bottom_height, const int bottom_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * bottom_height * bottom_width;
    const int bh = int(h * bottom_height / float(height));
    const int hw = int(w * bottom_width / float(width));
    top_data[index] = bottom_slice[bh * bottom_width + hw];
  }
}

template <typename Dtype>
__global__ void UpscaleBackward(const int nthreads,
    const Dtype* const top_diff, const int num, const int channels,
    const int height, const int width, const int top_height, const int top_width, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const Dtype* const top_slice =
        top_diff + (n * channels + c) * top_height * top_width;
    const int factor_h = int(ceil(top_height / float(height)));
    const int factor_w = int(ceil(top_width / float(width)));
    for (int i = h * factor_h; i < min((h + 1) * factor_h, top_height); i++) {
      for (int j = w * factor_w; j < min((w + 1) * factor_w, top_width); j++) {
        bottom_diff[index] += top_slice[i * top_width + j];
      }
    }
  }
}

template <typename Dtype>
void UpscaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  UpscaleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top[0]->num(), top[0]->channels(),
      top[0]->height(), top[0]->width(), bottom[0]->height(), bottom[0]->width(), top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void UpscaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // NOLINT_NEXT_LINE(whitespace/operators)
  UpscaleBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, bottom[0]->num(), bottom[0]->channels(),
    bottom[0]->height(), bottom[0]->width(), top[0]->height(), top[0]->width(), bottom_diff);

  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UpscaleLayer);


}  // namespace caffe

