#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/rpn_annotator_ohem_layer.hpp"

using std::max;
using std::min;

namespace caffe {
  template <typename Dtype>
  void RpnAnnotatorOHEMLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_loss = bottom[0]->cpu_data();
    const Dtype* bottom_labels = bottom[1]->cpu_data();
    const Dtype* bottom_bbox_loss_weights = bottom[2]->cpu_data();
    Dtype* top_labels = top[0]->mutable_cpu_data();
    Dtype* top_bbox_loss_weights = top[1]->mutable_cpu_data();
    caffe_set(top[0]->count(), Dtype(ignore_label_), top_labels);
    caffe_set(top[1]->count(), Dtype(0), top_bbox_loss_weights);

    int num_rpns_ = bottom[0]->count();

    // Find rois with max loss
    vector<int> sorted_idx(num_rpns_);
    for (int i = 0; i < num_rpns_; i++) {
      sorted_idx[i] = i;
    }
    std::sort(sorted_idx.begin(), sorted_idx.end(),
      [bottom_loss](int i1, int i2) {
        return bottom_loss[i1] > bottom_loss[i2];
    });

    // Generate output labels for scoring and loss_weights for bbox regression
    int number_pos_left = int(rpn_per_img_ * fg_fraction_ + 0.5);
    int number_neg_left = rpn_per_img_ - number_pos_left;
    for (int i = 0; i < num_rpns_; i++) {
      int index = sorted_idx[i];
      int s = index % (width_*height_);
      int n = index / (width_*height_);
      if (bottom_labels[index] == ignore_label_) {
        continue;
      } else if (bottom_labels[index] == positive_label_) {
        if (number_pos_left > 0) {
          number_pos_left--;
          top_labels[index] = bottom_labels[index];
          for (int j = 0; j < 4; j++) {
            int bbox_index = (n*4+j)*spatial_dim_+s;
            top_bbox_loss_weights[bbox_index] =
                bottom_bbox_loss_weights[bbox_index];
          }
        }
      } else if (bottom_labels[index] == negative_label_) {
        if (number_neg_left > 0) {
          number_neg_left--;
          top_labels[index] = bottom_labels[index];
          for (int j = 0; j < 4; j++) {
            int bbox_index = (n*4+j)*spatial_dim_+s;
            top_bbox_loss_weights[bbox_index] =
                bottom_bbox_loss_weights[bbox_index];
          }
        }
      }
    }
  }

  template <typename Dtype>
  void RpnAnnotatorOHEMLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    return;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(RpnAnnotatorOHEMLayer);

}  // namespace caffe
