#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/rpn_annotator_ohem_layer.hpp"

using std::max;
using std::min;

namespace caffe {
  template <typename Dtype>
  void RpnAnnotatorOHEMLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_loss = bottom[0]->cpu_data();
    const Dtype* bottom_labels = bottom[1]->cpu_data();
    const Dtype* bottom_bbox_loss_weights = bottom[2]->cpu_data();
    Dtype* top_labels = top[0]->mutable_cpu_data();
    Dtype* top_bbox_loss_weights = top[1]->mutable_cpu_data();
    caffe_set(top[0]->count(), Dtype(ignore_label_), top_labels);
    caffe_set(top[1]->count(), Dtype(0), top_bbox_loss_weights);

    int num_rpns_ = bottom[0]->count();

    // Find rois with max loss
    vector<int> sorted_idx(num_rpns_);
    for (int i = 0; i < num_rpns_; i++) {
      sorted_idx[i] = i;
    }
    std::sort(sorted_idx.begin(), sorted_idx.end(),
      [bottom_loss](int i1, int i2) {
        return bottom_loss[i1] > bottom_loss[i2];
    });

    // Generate output labels for scoring and loss_weights for bbox regression
    int number_left = rpn_per_img_;
    for (int i = 0; i < num_rpns_; i++) {
      int index = sorted_idx[i];
      int s = index % (width_*height_);
      int n = index / (width_*height_);
      if (bottom_labels[index] == ignore_label_) {
        continue;
      }
      if (number_left > 0) {
        number_left--;
        top_labels[index] = bottom_labels[index];
        for (int j = 0; j < 4; j++) {
          int bbox_index = (n*4+j)*spatial_dim_+s;
          top_bbox_loss_weights[bbox_index] =
            bottom_bbox_loss_weights[bbox_index];
        }
      }
    }
  }

  template <typename Dtype>
  void RpnAnnotatorOHEMLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    return;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(RpnAnnotatorOHEMLayer);

}  // namespace caffe
