#include "hip/hip_runtime.h"
// --------------------------------------------------------
// R-FCN
// --------------------------------------------------------

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/psroi_align_layer.hpp"
#include "caffe/util/gpu_util.cuh"

using std::max;
using std::min;

namespace caffe {

  template <typename Dtype>
  __global__ void PSROIAlignForward(
    const int nthreads,
    const Dtype* bottom_data,
    const Dtype spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois,
    const int output_dim,
    const int group_size,
    Dtype* top_data,
    int* mapping_channel) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)
      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int ctop = (index / pooled_width / pooled_height) % output_dim;
      int n = index / pooled_width / pooled_height / output_dim;

      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];
      Dtype roi_start_w = min(max(bottom_rois[1] * spatial_scale, Dtype(0)), static_cast<Dtype>(width - 1));
      Dtype roi_start_h = min(max(bottom_rois[2] * spatial_scale, Dtype(0)), static_cast<Dtype>(height - 1));
      Dtype roi_end_w = min(max(bottom_rois[3] * spatial_scale, Dtype(0)), static_cast<Dtype>(width - 1));
      Dtype roi_end_h = min(max(bottom_rois[4] * spatial_scale, Dtype(0)), static_cast<Dtype>(height - 1));

      Dtype roi_width = roi_end_w - roi_start_w;
      Dtype roi_height =roi_end_h - roi_start_h;

      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      Dtype hstart = static_cast<Dtype>(ph) * bin_size_h + roi_start_h;
      Dtype wstart = static_cast<Dtype>(pw)* bin_size_w + roi_start_w;
      Dtype hend = static_cast<Dtype>(ph + 1) * bin_size_h + roi_start_h;
      Dtype wend = static_cast<Dtype>(pw + 1) * bin_size_w + roi_start_w;

      bool is_empty = (hend <= hstart) || (wend <= wstart);

      int gw = pw;
      int gh = ph;
      int c = (ctop*group_size + gh)*group_size + gw;

      bottom_data += (roi_batch_ind * channels + c) * height * width;
      Dtype out_sum = 0;
      // Selecting four regular locations for bilinear interpolation
      for (Dtype h = hstart + bin_size_h / Dtype(4); h < hend; h += bin_size_h / Dtype(2)) {
        for (Dtype w = wstart + bin_size_w / Dtype(4); w < wend; w += bin_size_w / Dtype(2)) {
          int x_left = floor(w);
          int x_right = ceil(w);
          if (x_right == x_left) {
            x_right = x_left + 1;
          }
          int y_bottom = floor(h);
          int y_top = ceil(h);
          if (y_top == y_bottom) {
            y_top = y_bottom + 1;
          }

          int top_left_index = y_top * width + x_left;
          int top_right_index = y_top * width + x_right;
          int bottom_left_index = y_bottom * width + x_left;
          int bottom_right_index = y_bottom * width + x_right;

          Dtype val = 0;
          val += (1 - w + x_left) * (1 - y_top + h) * bottom_data[top_left_index];
          val += (1 - x_right + w) * (1 - y_top + h) * bottom_data[top_right_index];
          val += (1 - w + x_left) * (1 - h + y_bottom) * bottom_data[bottom_left_index];
          val += (1 - x_right + w) * (1 - h + y_bottom) * bottom_data[bottom_right_index];

          out_sum += val;
        }
      }

      top_data[index] = is_empty? 0. : out_sum / 4;
      mapping_channel[index] = c;
    }
  }

  template <typename Dtype>
  void PSROIAlignLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* mapping_channel_ptr = mapping_channel_.mutable_gpu_data();
    int count = top[0]->count();
    caffe_gpu_set(count, Dtype(0), top_data);
    caffe_gpu_set(count, -1, mapping_channel_ptr);
    // NOLINT_NEXT_LINE(whitespace/operators)
    PSROIAlignForward<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, bottom_data, spatial_scale_,
      channels_, height_, width_, pooled_height_,
      pooled_width_, bottom_rois, output_dim_, group_size_,
      top_data, mapping_channel_ptr);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  __global__ void PSROIAlignBackwardAtomic(
    const int nthreads,
    const Dtype* top_diff,
    const int* mapping_channel,
    const int num_rois,
    const Dtype spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int output_dim,
    Dtype* bottom_diff,
    const Dtype* bottom_rois) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)
      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int n = index / pooled_width / pooled_height / output_dim;

      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];
      Dtype roi_start_w = min(max(bottom_rois[1] * spatial_scale, Dtype(0)), static_cast<Dtype>(width - 1));
      Dtype roi_start_h = min(max(bottom_rois[2] * spatial_scale, Dtype(0)), static_cast<Dtype>(height - 1));
      Dtype roi_end_w = min(max(bottom_rois[3] * spatial_scale, Dtype(0)), static_cast<Dtype>(width - 1));
      Dtype roi_end_h = min(max(bottom_rois[4] * spatial_scale, Dtype(0)), static_cast<Dtype>(height - 1));

      Dtype roi_width = roi_end_w - roi_start_w;
      Dtype roi_height = roi_end_h - roi_start_h;

      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      Dtype hstart = static_cast<Dtype>(ph) * bin_size_h + roi_start_h;
      Dtype wstart = static_cast<Dtype>(pw)* bin_size_w + roi_start_w;
      Dtype hend = static_cast<Dtype>(ph + 1) * bin_size_h + roi_start_h;
      Dtype wend = static_cast<Dtype>(pw + 1) * bin_size_w + roi_start_w;

      bool is_empty = (hend <= hstart) || (wend <= wstart);

      // Compute c at bottom
      int c = mapping_channel[index];
      Dtype* offset_bottom_diff = bottom_diff +
        (roi_batch_ind * channels + c) * height * width;
      Dtype diff_val = is_empty ? 0. : top_diff[index] / 4;
      // Selecting four regular locations for bilinear interpolation
      for (Dtype h = hstart + bin_size_h / Dtype(4); h < hend; h += bin_size_h / Dtype(2)) {
        for (Dtype w = wstart + bin_size_w / Dtype(4); w < wend; w += bin_size_w / Dtype(2)) {
          int x_left = floor(w);
          int x_right = ceil(w);
          if (x_right == x_left) {
            x_right = x_left + 1;
          }
          int y_bottom = floor(h);
          int y_top = ceil(h);
          if (y_top == y_bottom) {
            y_top = y_bottom + 1;
          }

          int top_left_index = y_top * width + x_left;
          int top_right_index = y_top * width + x_right;
          int bottom_left_index = y_bottom * width + x_left;
          int bottom_right_index = y_bottom * width + x_right;

          caffe_gpu_atomic_add(diff_val * (x_right - w) * (h - y_bottom), offset_bottom_diff + top_left_index);
          caffe_gpu_atomic_add(diff_val * (w - x_left) * (h - y_bottom), offset_bottom_diff + top_right_index);
          caffe_gpu_atomic_add(diff_val * (x_right - w) * (y_top - h), offset_bottom_diff + bottom_left_index);
          caffe_gpu_atomic_add(diff_val * (w - x_left) * (y_top - h), offset_bottom_diff + bottom_right_index);
        }
      }
    }
  }

  template <typename Dtype>
  void PSROIAlignLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
      return;
    }

    const Dtype* bottom_rois = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    const int* mapping_channel_ptr = mapping_channel_.gpu_data();
    caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
    caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);
    const int count = top[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    PSROIAlignBackwardAtomic<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, top_diff, mapping_channel_ptr,
      top[0]->num(), spatial_scale_, channels_, height_, width_,
      pooled_height_, pooled_width_, output_dim_, bottom_diff,
      bottom_rois);
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(PSROIAlignLayer);

}  // namespace caffe
