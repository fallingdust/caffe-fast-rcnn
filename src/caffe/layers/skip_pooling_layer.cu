#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/skip_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SkipPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % (channels * 4);
    const int n = index / pooled_width / pooled_height / (channels * 4);
    int hstart = ph * 2;
    int wstart = pw * 2;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c / 4) * height * width;
    int h_index = hstart + c % 4 / 2;
    int w_index = wstart + c % 4 % 2;
    top_data[index] = (h_index >= height || w_index >= width) ? Dtype(0) : bottom_slice[h_index * width + w_index];
  }
}


template <typename Dtype>
void SkipPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SkipPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), channels_,
      height_, width_, pooled_height_, pooled_width_, top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void SkipPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}


INSTANTIATE_LAYER_GPU_FUNCS(SkipPoolingLayer);


}  // namespace caffe
