#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/skip_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SkipPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % (channels * 4);
    const int n = index / pooled_width / pooled_height / (channels * 4);
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    const int hend = min(hstart + kernel_h, height);
    const int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    top_data[index] = Dtype(0);
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c / 4) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        top_data[index] = bottom_slice[h * width + w];
      }
    }
  }
}


template <typename Dtype>
void PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SkipPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), channels_,
      height_, width_, pooled_height_, pooled_width_, 2,
      2, 2, 2, 0, 0, top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void PoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}


INSTANTIATE_LAYER_GPU_FUNCS(SkipPoolingLayer);


}  // namespace caffe
