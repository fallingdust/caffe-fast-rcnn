#include "hip/hip_runtime.h"

#include <cfloat>

#include "caffe/layers/roi_align_layer.hpp"

using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
__global__ void ROIAlignForward(const int nthreads, const Dtype* bottom_data,
    const Dtype spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois, Dtype* top_data, Dtype* argmax_data_x, Dtype* argmax_data_y) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = bottom_rois[0];
    Dtype roi_start_w = bottom_rois[1] * spatial_scale;
    Dtype roi_start_h = bottom_rois[2] * spatial_scale;
    Dtype roi_end_w = bottom_rois[3] * spatial_scale;
    Dtype roi_end_h = bottom_rois[4] * spatial_scale;

    Dtype roi_width = roi_end_w - roi_start_w;
    Dtype roi_height = roi_end_h - roi_start_h;
    Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

    Dtype hstart = static_cast<Dtype>(ph) * bin_size_h;
    Dtype wstart = static_cast<Dtype>(pw) * bin_size_w;
    Dtype hend = static_cast<Dtype>(ph + 1) * bin_size_h;
    Dtype wend = static_cast<Dtype>(pw + 1) * bin_size_w;

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, Dtype(0)), static_cast<Dtype>(height - 1));
    hend = min(max(hend + roi_start_h, Dtype(0)), static_cast<Dtype>(height - 1));
    wstart = min(max(wstart + roi_start_w, Dtype(0)), static_cast<Dtype>(width - 1));
    wend = min(max(wend + roi_start_w, Dtype(0)), static_cast<Dtype>(width - 1));
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    Dtype maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    Dtype maxidx_x = -1;
    Dtype maxidx_y = -1;
    bottom_data += (roi_batch_ind * channels + c) * height * width;
    // Selecting the center locations for bilinear interpolation
    Dtype h = hstart + bin_size_h / Dtype(2);
    Dtype w = wstart + bin_size_w / Dtype(2);
    int x_left = floor(w);
    int x_right = ceil(w);
    if (x_right == x_left) {
      x_right = x_left + 1;
    }
    int y_bottom = floor(h);
    int y_top = ceil(h);
    if (y_top == y_bottom) {
      y_top = y_bottom + 1;
    }

    int top_left_index = y_top * width + x_left;
    int top_right_index = y_top * width + x_right;
    int bottom_left_index = y_bottom * width + x_left;
    int bottom_right_index = y_bottom * width + x_right;

    Dtype val = 0;
    val += (1 - w + x_left) * (1 - y_top + h) * bottom_data[top_left_index];
    val += (1 - x_right + w) * (1 - y_top + h) * bottom_data[top_right_index];
    val += (1 - w + x_left) * (1 - h + y_bottom) * bottom_data[bottom_left_index];
    val += (1 - x_right + w) * (1 - h + y_bottom) * bottom_data[bottom_right_index];

    if (val > maxval) {
      maxval = val;
      maxidx_x = w;
      maxidx_y = h;
    }
    top_data[index] = maxval;
    argmax_data_x[index] = maxidx_x;
    argmax_data_y[index] = maxidx_y;
  }
}

template <typename Dtype>
void ROIAlignLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* argmax_data_x = max_idx_x_.mutable_gpu_data();
  Dtype* argmax_data_y = max_idx_y_.mutable_gpu_data();
  int count = top[0]->count();
  if (bottom[1]->num() == 0) {
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  ROIAlignForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, spatial_scale_, channels_, height_, width_,
      pooled_height_, pooled_width_, bottom_rois, top_data, argmax_data_x, argmax_data_y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ROIAlignBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* argmax_data_x, const Dtype* argmax_data_y, const int num_rois, const Dtype spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, Dtype* bottom_diff,
    const Dtype* bottom_rois) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, h, w) coords in bottom data
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    Dtype gradient = 0;
    // Accumulate gradient over all ROIs that pooled this element
    for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
      const Dtype* offset_bottom_rois = bottom_rois + roi_n * 5;
      int roi_batch_ind = offset_bottom_rois[0];
      // Skip if ROI's batch index doesn't match n
      if (n != roi_batch_ind) {
        continue;
      }

      int roi_start_w = floor(offset_bottom_rois[1] * spatial_scale);
      int roi_start_h = floor(offset_bottom_rois[2] * spatial_scale);
      int roi_end_w = ceil(offset_bottom_rois[3] * spatial_scale);
      int roi_end_h = ceil(offset_bottom_rois[4] * spatial_scale);

      // Skip if ROI doesn't include (h, w)
      const bool in_roi = (w >= roi_start_w && w <= roi_end_w &&
                           h >= roi_start_h && h <= roi_end_h);
      if (!in_roi) {
        continue;
      }

      int offset = (roi_n * channels + c) * pooled_height * pooled_width;
      const Dtype* offset_top_diff = top_diff + offset;
      const Dtype* offset_argmax_data_x = argmax_data_x + offset;
      const Dtype* offset_argmax_data_y = argmax_data_y + offset;

      for (int ph = 0; ph < pooled_height; ++ph) {
        for (int pw = 0; pw < pooled_width; ++pw) {
          int pindex = ph * pooled_width + pw;
          Dtype max_x = offset_argmax_data_x[pindex];
          Dtype max_y = offset_argmax_data_y[pindex];

          int x_left = floor(max_x);
          int x_right = ceil(max_x);
          if (x_right == x_left) {
            x_right = x_left + 1;
          }
          int y_bottom = floor(max_y);
          int y_top = ceil(max_y);
          if (y_top == y_bottom) {
            y_top = y_bottom + 1;
          }

          if (x_left == w && y_top == h)
            gradient += (1 - max_x + x_left) * (1 - y_top + max_y) * offset_top_diff[pindex];
          else if (x_left == w && y_bottom == h)
            gradient += (1 - max_x + x_left) * (1 - max_y + y_bottom) * offset_top_diff[pindex];
          else if (x_right == w && y_top == h)
            gradient += (1 - x_right + max_x) * (1 - y_top + max_y) * offset_top_diff[pindex];
          else if (x_right == w && y_bottom == h)
            gradient += (1 - x_right + max_x) * (1 - max_y + y_bottom) * offset_top_diff[pindex];
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void ROIAlignLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0] || top[0]->num() == 0) {
    return;
  }
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  const Dtype* argmax_data_x = max_idx_x_.gpu_data();
  const Dtype* argmax_data_y = max_idx_y_.gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ROIAlignBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, argmax_data_x, argmax_data_y, top[0]->num(), spatial_scale_, channels_,
      height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(ROIAlignLayer);

}  // namespace caffe
